#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

// simple vector addition kernel
__global__
void vecAddKernel(float* A, float* B, float* C, int n)
{
	// compute index from block and thread indices
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	// check boundary condition
	if (index < n)
		C[index] = A[index] + B[index];
}

// add to vectors h_A and h_B to get h_C of size n on GPU
void vecAdd(float* h_A, float* h_B, float* h_C, int n)
{
	const int THREAD_SIZE = 1024;

	// for cudaMalloc and cudaMemcpy
	int size = n * sizeof(float);

	// device variables
	float* d_A;
	float* d_B;
	float* d_C;

	// malloc vector A
	hipMalloc((void**) &d_A, size);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	// malloc vector B
	hipMalloc((void**) &d_B, size);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// malloc vector C
	hipMalloc((void**) &d_C, size);

	// run vector addition kernel
	dim3 DimGrid((n-1)/THREAD_SIZE + 1, 1, 1);
	dim3 DimBlock(THREAD_SIZE, 1, 1);
	vecAddKernel<<<DimGrid,DimBlock>>>(d_A, d_B, d_C, n);

	// copy contents
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// free memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

// This program adds two vectors together using CUDA.
int main(int argc, char* argv[])
{
	const int VECTOR_SIZE = 10000;

	// create addend vector A
	float* A = new float[VECTOR_SIZE];
	for (int i = 0; i < VECTOR_SIZE; i++)
		A[i] = 2*i;

	// create addend vector B
	float* B = new float[VECTOR_SIZE];
	for (int i = 0; i < VECTOR_SIZE; i++)
		B[i] = 3*i;

	// create sum vector C
	float* C = new float[VECTOR_SIZE];
	for (int i = 0; i < VECTOR_SIZE; i++)
		C[i] = 0;

	// perform vector addition with CUDA
	vecAdd(A, B, C, VECTOR_SIZE);

	// print result
	for (int i = 0; i < VECTOR_SIZE; i++)
		cout << "C[" << i <<  "]=" << C[i] << endl;

	// free memory
	delete[] A;
	delete[] B;
	delete[] C;

	return 0;
}